
#include <hip/hip_runtime.h>
#include"stdio.h"
#include <string.h>
#include <stdlib.h>
#include <time.h>
#include"math.h"
#include <ctype.h>
#include <assert.h>

/*this code shows that starting from the least cost initial solution through NN and performing 2 opt only on the best initial solution 
may or may not reach a better solution than IHC.c (varies for instances - gives improvement for 300 and 1000 instances, but not for
instance size 100)*/

/* Euclidean distance calculation */
__host__ __device__ long distD(int i,int j,float *x,float*y)
{
	float dx=x[i]-x[j];
	float dy=y[i]-y[j]; 
	return(sqrtf( (dx*dx) + (dy*dy) ));
}

// long distD(int i,int j,float *x,float*y)
// {
// 	float dx=x[i]-x[j];
// 	float dy=y[i]-y[j]; 
// 	return(sqrtf( (dx*dx) + (dy*dy) ));
// }

// __device__ void routeChecker(long N,int *r)
// {
// 	int *v,i,flag=0;
// 	v=(int*)calloc(N,sizeof(int));	

// 	for(i=0;i<N;i++)
// 		v[r[i]]++;
// 	for(i=0;i<N;i++)
// 	{
// 		if(v[i] != 1 )
// 		{
// 			flag=1;
// 			printf("breaking at %d",i);
// 			break;
// 		}
// 	}
// 	if(flag==1)
// 		printf("\nroute is not valid");
// 	// else
// 	// 	printf("\nroute is valid");
// }

/* Initial solution construction using NN */
__global__ void nn_init(int *route,long cities,float *posx,float*posy,int *visited,long *dst)
{
	int id = threadIdx.x+blockIdx.x*blockDim.x;
	if(id<cities)
	{
		dst[id]=0;
		int start_index=id;
		route[start_index*cities+0]=start_index;
		int k=1,i=start_index,j;
		float min;
		int minj,mini,count=1,flag=0;
		// long dst=0;
		// int *visited=(int*)calloc(cities,sizeof(int));
		visited[start_index*cities+start_index]=1;
		while(count!=cities)
		{
			flag=0;
			for(j=0;j<cities;j++)
			{
				if(i!=j && !visited[start_index*cities+j])
				{
					min=distD(i,j,posx,posy);
					minj=j;
					break;	
				}
			}

			for(j=minj+1;j<cities;j++)
			{
				
				if( !visited[start_index*cities+j])
				{
					if(min>distD(i,j,posx,posy))
					{
						min=distD(i,j,posx,posy);
						mini=j;
						flag=1;				
					}
				}
			}
			if(flag==0)
				i=minj;
			else
				i=mini;
			dst[id]+=min;
			route[start_index*cities+k++]=i;
			visited[start_index*cities+i]=1;
			count++;
		}
		// free(visited);
		dst[id]+=distD(route[start_index*cities+0],route[start_index*cities+cities-1],posx,posy);
		// routeChecker(cities, route);
	}

	
}


/* Arrange coordinate in initial solution's order*/
void setCoord(int *r,float *posx,float *posy,float *px,float *py,long cities)
{
	int i;
	for(i=0;i<cities;i++)
	{
		px[i]=posx[r[i]];
		py[i]=posy[r[i]];
	}
}

long distH(float *px,float *py,long cit)
{
	float dx,dy;
	long cost=0;
	int i;
	for(i=0;i<(cit-1);i++)
	{
		dx=px[i]-px[i+1];
		dy=py[i]-py[i+1]; 
		cost+=sqrtf( (dx*dx) + (dy*dy) );
	}
	dx=px[i]-px[0];
	dy=py[i]-py[0]; 
	cost+=sqrtf( (dx*dx) + (dy*dy) );
	return cost;

}

int minn(int a,int b)
{
	if(a<b)
	{
		return a;
	}
	return b;
}

/*A kenel function that finds a minimal weighted neighbor using TPR mapping strategy*/
__global__ void tsp_tpr(float *pox,float *poy,long initcost,unsigned long long *dst_tid,long cit)
{

	long id,j;
	register long change,mincost=initcost,cost;
	long i=threadIdx.x+blockIdx.x*blockDim.x;
	if(i < cit)
	{
		
		for(j=i+1;j<cit;j++)
		{
			change = 0; cost=initcost;
			change=distD(i,j,pox,poy)+distD((i+1)%cit,(j+1)%cit,pox,poy)-distD(i,(i+1)%cit,pox,poy)-distD(j,(j+1)%cit,pox,poy);
			cost+=change;	
			if(cost < mincost)
			{
				mincost = cost;
				id = i * (cit-1)+(j-1)-i*(i+1)/2;	
			}	 

		}
		if(mincost < initcost)
			 atomicMin(dst_tid, ((unsigned long long)mincost << 32) | id);

	}
	
}

/*A kenel function that finds a minimal weighted neighbor using TPRED mapping strategy*/
__global__ void tsp_tpred(float *pox,float *poy,long initcost,unsigned long long *dst_tid,long cit,long itr)
{
	long id,j,k;
	register long change,mincost=initcost,cost;
	long i=threadIdx.x+blockIdx.x*blockDim.x;
	if(i < cit)
	{
		
		for(k=0;k<itr;k++)
		{
			change = 0; cost=initcost;
			j=(i+1+k)%cit;
			change=distD(i,j,pox,poy)+distD((i+1)%cit,(j+1)%cit,pox,poy)-distD(i,(i+1)%cit,pox,poy)-distD(j,(j+1)%cit,pox,poy);
			cost+=change;	
			if(cost < mincost)
			{
				mincost = cost;
				if(i < j)
					id = i * (cit-1)+(j-1)-i*(i+1)/2;	
				else
					id = j * (cit-1)+(i-1)-j*(j+1)/2;	

			}	 

		}
		if(mincost < initcost)
			 atomicMin(dst_tid, ((unsigned long long)mincost << 32) | id);
	}
}

/*A kenel function that finds a minimal weighted neighbor using TPRC mapping strategy*/
__global__ void tsp_tprc(float *pox,float *poy,long initcost,unsigned long long *dst_tid,long cit)
{

	long id;
	long change,cost;
	long i=threadIdx.x+blockIdx.x*blockDim.x;
	long j=threadIdx.y+blockIdx.y*blockDim.y;
	if(i < cit && j < cit && i < j)
	{
		
			change = 0; cost = initcost;
			change=distD(i,j,pox,poy)+distD((i+1)%cit,(j+1)%cit,pox,poy)-distD(i,(i+1)%cit,pox,poy)-distD(j,(j+1)%cit,pox,poy);
			cost+=change;	
			if(change < 0)
			{
				id = i * (cit - 1) + (j - 1) - i * (i + 1) / 2;	
				atomicMin(dst_tid, ((unsigned long long)cost << 32) | id);
			}	 

	}
	
}

/*A kenel function that finds a minimal weighted neighbor using TPN mapping strategy*/
__global__ void tsp_tpn(float *pox,float *poy,long cost,unsigned long long *dst_tid,long cit,long sol)
{

	long i,j;
	register long change=0;
	int id=threadIdx.x+blockIdx.x*blockDim.x;
	if(id<sol)
	{
		
		i=cit-2-floorf(((int)__dsqrt_rn(8*(sol-id-1)+1)-1)/2);
		j=id-i*(cit-1)+(i*(i+1)/2)+1;
		change=distD(i,j,pox,poy)+distD((i+1)%cit,(j+1)%cit,pox,poy)-distD(i,(i+1)%cit,pox,poy)-distD(j,(j+1)%cit,pox,poy);
		cost+=change;	
		if(change < 0)
			 atomicMin(dst_tid, ((unsigned long long)cost << 32) | id);
		
	}
	
}

/* At each IHC steps, XY coordinates are arranged using next initial solution's order*/
void twoOpt(long x,long y,float *pox,float *poy)
{
	float *tmp_x,*tmp_y;
	int i,j;
	tmp_x=(float*)malloc(sizeof(float)*(y-x));	
	tmp_y=(float*)malloc(sizeof(float)*(y-x));
	for(j=0,i=y;i>x;i--,j++)
	{
		tmp_x[j]=pox[i];
		tmp_y[j]=poy[i];
	}
	for(j=0,i=x+1;i<=y;i++,j++)
	{
		pox[i]=tmp_x[j];
		poy[i]=tmp_y[j];
	}
	free(tmp_x);
	free(tmp_y);

}

int main(int argc, char *argv[])
{
	int ch, cnt, in1;
	float in2, in3;
	FILE *f;
	float *posx, *posy;
	float *px, *py,tm;
	char str[256];  
	int *r;
	long sol,d,cities,no_pairs,tid=0;
	int i,j,intl,count;
	
	clock_t start,end,start1,end1;

	f = fopen(argv[1], "r");
	if (f == NULL) {fprintf(stderr, "could not open file \n");  exit(-1);}

	ch = getc(f);  while ((ch != EOF) && (ch != '\n')) ch = getc(f);
	ch = getc(f);  while ((ch != EOF) && (ch != '\n')) ch = getc(f);
	ch = getc(f);  while ((ch != EOF) && (ch != '\n')) ch = getc(f);

	ch = getc(f);  while ((ch != EOF) && (ch != ':')) ch = getc(f);
	fscanf(f, "%s\n", str);
	cities = atoi(str);
	if (cities <= 2) {fprintf(stderr, "only %ld cities\n", cities);  exit(-1);}

	sol=cities*(cities-1)/2;
	posx = (float *)malloc(sizeof(float) * cities);  if (posx == NULL) {fprintf(stderr, "cannot allocate posx\n");  exit(-1);}
	posy = (float *)malloc(sizeof(float) * cities);  if (posy == NULL) {fprintf(stderr, "cannot allocate posy\n");  exit(-1);}
	px = (float *)malloc(sizeof(float) * cities);  if (px == NULL) {fprintf(stderr, "cannot allocate posx\n");  exit(-1);}
	py = (float *)malloc(sizeof(float) * cities);  if (py == NULL) {fprintf(stderr, "cannot allocate posy\n");  exit(-1);}
	
	r = (int *)malloc(sizeof(int) * cities);
	ch = getc(f);  while ((ch != EOF) && (ch != '\n')) ch = getc(f);
	fscanf(f, "%s\n", str);
	if (strcmp(str, "NODE_COORD_SECTION") != 0) {fprintf(stderr, "wrong file format\n");  exit(-1);}

	cnt = 0;

	while (fscanf(f, "%d %f %f\n", &in1, &in2, &in3)) 
	{
		posx[cnt] = in2;
		posy[cnt] = in3;
		cnt++;
		if (cnt > cities) {fprintf(stderr, "input too long\n");  exit(-1);}
		if (cnt != in1) {fprintf(stderr, "input line mismatch: expected %d instead of %d\n", cnt, in1);  exit(-1);}
	}

	if (cnt != cities) {fprintf(stderr, "read %d instead of %ld cities\n", cnt, cities);  exit(-1);}
	fscanf(f, "%s", str);
	if (strcmp(str, "EOF") != 0) {fprintf(stderr, "didn't see 'EOF' at end of file\n");  exit(-1);}

	// int dst_final=INT_MAX;
	// int count_final;
	// int best_initial_dst=INT_MAX;
	// double best_initial_time;
	// int best_start_city;
    // int*best_initial_route = (int *)malloc(sizeof(int) * cities);

	// if(cities>1000)
	// {
	// 	printf("too many cities, code does not support yet");
	// 	return 0;
	// }

	if(cities<1)
	{
		printf("too less cities");
		return 0;
	}

	long *dst;
	int *visited;
	long *dst_host;
	int *r_device;

	if(hipSuccess!=hipMalloc((void**)&dst,sizeof(long)*cities))
	printf("\nCan't allocate memory for dst in device");

	dst_host=(long*)malloc(sizeof(long)*(cities));	

	if(hipSuccess!=hipMalloc((void**)&visited,sizeof(int)*(cities*cities)))
	printf("\nCan't allocate memory for visited in device");

	if(hipSuccess!=hipMalloc((void**)&r_device,sizeof(int)*(cities*cities)))
	printf("\nCan't allocate memory for r i.e route in device");

	float *d_posx, *d_posy;

	if(hipSuccess!=hipMalloc((void**)&d_posx,sizeof(float)*cities))
	printf("\nCan't allocate memory for coordinate x on GPU");
	if(hipSuccess!=hipMalloc((void**)&d_posy,sizeof(float)*cities))
	printf("\nCan't allocate memory for coordinate y on GPU");

	if(hipSuccess!=hipMemcpy(d_posx,posx,sizeof(float)*cities,hipMemcpyHostToDevice))
	printf("\nCan't transfer px on GPU");
	if(hipSuccess!=hipMemcpy(d_posy,posy,sizeof(float)*cities,hipMemcpyHostToDevice))
	printf("\nCan't transfer py on GPU");

	start = clock();
	
	/*Calling NN algo for initial solution creation*/
	nn_init<<<(cities-1/1024)+1,minn(cities,1024)>>>(r_device,cities,d_posx,d_posy,visited,dst);

	if(hipSuccess!=hipMemcpy(dst_host,dst,sizeof(long)*cities,hipMemcpyDeviceToHost))
	printf("\nCan't transfer dst values back to CPU");

	end = clock();

	tm = ((double) (end - start)) / CLOCKS_PER_SEC;

	long best_initial_dst=INT_MAX;
	int best_start_city;

	for(int i=0;i<cities;i++)
	{
		// printf("\nindex : %d , value at index : %ld",i,dst_host[i]);
		if(dst_host[i]<best_initial_dst)
		{
			best_initial_dst=dst_host[i];
			best_start_city=i;
		}
	}

	int *req_r=r_device+best_start_city*cities; //move only the route which corresponds to minimum initial dst

	if(hipSuccess!=hipMemcpy(r,req_r,sizeof(int)*cities,hipMemcpyDeviceToHost))
	printf("\nCan't transfer best route values back to CPU");

	// for(int i=0;i<cities;i++)
	// {
	// 	printf("\n%d",r[i]);
	// }

    setCoord(r,posx,posy,px,py,cities);

	int blk,thrd;
	unsigned long long *d_dst_tid;
	long dst2=best_initial_dst;
	long x,y;

	start1 = clock();
	count = 1;
	unsigned long long dst_tid = (((long)dst2+1) << 32) -1;
        unsigned long long dtid;
	long itr=floor(cities/2);
	int nx, ny;
	if(cities <= 32)
	{
		blk = 1 ;
		nx = cities;
		ny = cities;
	}
	else
	{
		blk = (cities - 1) / 32 + 1;
		nx = 32;
		ny = 32;
	}
	dim3 thrds (nx,ny);
	dim3 blks (blk,blk);
	if(hipSuccess!=hipMalloc((void**)&d_posx,sizeof(float)*cities))
	printf("\nCan't allocate memory for coordinate x on GPU");
	if(hipSuccess!=hipMalloc((void**)&d_posy,sizeof(float)*cities))
	printf("\nCan't allocate memory for coordinate y on GPU");
	if(hipSuccess!=hipMalloc((void**)&d_dst_tid,sizeof(unsigned long long)))
	printf("\nCan't allocate memory for dst_tid on GPU");
    	if(hipSuccess!=hipMemcpy(d_dst_tid,&dst_tid,sizeof(unsigned long long),hipMemcpyHostToDevice))
	printf("\nCan't transfer dst_tid on GPU");
	if(hipSuccess!=hipMemcpy(d_posx,px,sizeof(float)*cities,hipMemcpyHostToDevice))
	printf("\nCan't transfer px on GPU");
	if(hipSuccess!=hipMemcpy(d_posy,py,sizeof(float)*cities,hipMemcpyHostToDevice))
	printf("\nCan't transfer py on GPU");

	int strat;	
	printf("\n Choose a CUDA thread mapping strategy\n1.TPR\n2.TPRED\n3.TPRC\n4.TPN\n");
	scanf("%d",&strat);
	switch(strat)
	{
		case 1:

			if(cities<=1024)
			{
				blk=1;
				thrd=cities;
			}
			else
			{
				blk=(cities-1)/1024+1;
				thrd=1024;
			}
			
			tsp_tpr<<<blk,thrd>>>(d_posx,d_posy,dst2,d_dst_tid,cities);
			
			if(hipSuccess!=hipMemcpy(&dtid,d_dst_tid,sizeof(unsigned long long),hipMemcpyDeviceToHost))
			printf("\nCan't transfer minimal cost back to CPU");

			d = dtid >> 32;
			
			while( d < dst2 )
			{
				dst2=d;
				tid = dtid & ((1ull<<32)-1); 
				x=cities-2-floor((sqrt(8*(sol-tid-1)+1)-1)/2);
				y=tid-x*(cities-1)+(x*(x+1)/2)+1;
				twoOpt(x,y,px,py);
				if(hipSuccess!=hipMemcpy(d_posx,px,sizeof(float)*cities,hipMemcpyHostToDevice))
				printf("\nCan't transfer px on GPU");
				if(hipSuccess!=hipMemcpy(d_posy,py,sizeof(float)*cities,hipMemcpyHostToDevice))
				printf("\nCan't transfer py on GPU");
				unsigned long long dst_tid = (((long)dst2+1) << 32) -1;
				if(hipSuccess!=hipMemcpy(d_dst_tid,&dst_tid,sizeof(unsigned long long),hipMemcpyHostToDevice))
				printf("\nCan't transfer dst_tid on GPU");

				tsp_tpr<<<blk,thrd>>>(d_posx,d_posy,dst2,d_dst_tid,cities);
				if(hipSuccess!=hipMemcpy(&dtid,d_dst_tid,sizeof(unsigned long long),hipMemcpyDeviceToHost))
				printf("\nCan't transfer minimal cost back to CPU");
			  	d = dtid >> 32;
				count++;
			}
		break;
		case 2:
			
			if(cities<1024)
			{
				blk=1;
				thrd=cities;
			}
			else
			{
				blk=(cities-1)/1024+1;
				thrd=1024;
			}	

			tsp_tpred<<<blk,thrd>>>(d_posx,d_posy,dst2,d_dst_tid,cities,itr);
			
			if(hipSuccess!=hipMemcpy(&dtid,d_dst_tid,sizeof(unsigned long long),hipMemcpyDeviceToHost))
			printf("\nCan't transfer minimal cost back to CPU");

			d = dtid >> 32;
			
			while( d < dst2 )
			{

				dst2=d;
				tid = dtid & ((1ull<<32)-1); 
				x=cities-2-floor((sqrt(8*(sol-tid-1)+1)-1)/2);
				y=tid-x*(cities-1)+(x*(x+1)/2)+1;
				twoOpt(x,y,px,py);
				if(hipSuccess!=hipMemcpy(d_posx,px,sizeof(float)*cities,hipMemcpyHostToDevice))
				printf("\nCan't transfer px on GPU");
				if(hipSuccess!=hipMemcpy(d_posy,py,sizeof(float)*cities,hipMemcpyHostToDevice))
				printf("\nCan't transfer py on GPU");
				unsigned long long dst_tid = (((long)dst2+1) << 32) -1;
				if(hipSuccess!=hipMemcpy(d_dst_tid,&dst_tid,sizeof(unsigned long long),hipMemcpyHostToDevice))
				printf("\nCan't transfer dst_tid on GPU");

				tsp_tpred<<<blk,thrd>>>(d_posx,d_posy,dst2,d_dst_tid,cities,itr);
				
				if(hipSuccess!=hipMemcpy(&dtid,d_dst_tid,sizeof(unsigned long long),hipMemcpyDeviceToHost))
				printf("\nCan't transfer minimal cost back to CPU");
			  	d = dtid >> 32;
				count++;
			}
		break;
		case 3:
			
			tsp_tprc<<<blks,thrds>>>(d_posx,d_posy,dst2,d_dst_tid,cities);
	
			if(hipSuccess!=hipMemcpy(&dtid,d_dst_tid,sizeof(unsigned long long),hipMemcpyDeviceToHost))
			printf("\nCan't transfer minimal cost back to CPU");
		  	d = dtid >> 32;
			
			while( d < dst2 )
			{
				dst2=d;
				tid = dtid & ((1ull<<32)-1); 
				x=cities-2-floor((sqrt(8*(sol-tid-1)+1)-1)/2);
				y=tid-x*(cities-1)+(x*(x+1)/2)+1;
				twoOpt(x,y,px,py);
				if(hipSuccess!=hipMemcpy(d_posx,px,sizeof(float)*cities,hipMemcpyHostToDevice))
				printf("\nCan't transfer px on GPU");
				if(hipSuccess!=hipMemcpy(d_posy,py,sizeof(float)*cities,hipMemcpyHostToDevice))
				printf("\nCan't transfer py on GPU");
				unsigned long long dst_tid = (((long)dst2+1) << 32) -1;
				if(hipSuccess!=hipMemcpy(d_dst_tid,&dst_tid,sizeof(unsigned long long),hipMemcpyHostToDevice))
				printf("\nCan't transfer dst_tid on GPU");

				tsp_tprc<<<blks,thrds>>>(d_posx,d_posy,dst2,d_dst_tid,cities);
				if(hipSuccess!=hipMemcpy(&dtid,d_dst_tid,sizeof(unsigned long long),hipMemcpyDeviceToHost))
				printf("\nCan't transfer minimal cost back to CPU");
			  	d = dtid >> 32;
				count++;
			}
		break;
		case 4:
			if(sol < 1024)
			{
				blk=1;
				thrd=sol;
			}
			else
			{
				blk=(sol-1)/1024+1;
				thrd=1024;
			}

			tsp_tpn<<<blk,thrd>>>(d_posx,d_posy,dst2,d_dst_tid,cities,sol);

			if(hipSuccess!=hipMemcpy(&dtid,d_dst_tid,sizeof(unsigned long long),hipMemcpyDeviceToHost))
			printf("\nCan't transfer minimal cost back to CPU");
			d = dtid >> 32;
			
			while( d < dst2 )
			{
				dst2=d;
				tid = dtid & ((1ull<<32)-1); 
				x=cities-2-floor((sqrt(8*(sol-tid-1)+1)-1)/2);
				y=tid-x*(cities-1)+(x*(x+1)/2)+1;
				twoOpt(x,y,px,py);
				if(hipSuccess!=hipMemcpy(d_posx,px,sizeof(float)*cities,hipMemcpyHostToDevice))
				printf("\nCan't transfer px on GPU");
				if(hipSuccess!=hipMemcpy(d_posy,py,sizeof(float)*cities,hipMemcpyHostToDevice))
				printf("\nCan't transfer py on GPU");
				unsigned long long dst_tid = (((long)dst2+1) << 32) -1;
				if(hipSuccess!=hipMemcpy(d_dst_tid,&dst_tid,sizeof(unsigned long long),hipMemcpyHostToDevice))
				printf("\nCan't transfer dst_tid on GPU");

				tsp_tpn<<<blk,thrd>>>(d_posx,d_posy,dst2,d_dst_tid,cities,sol);

				if(hipSuccess!=hipMemcpy(&dtid,d_dst_tid,sizeof(unsigned long long),hipMemcpyDeviceToHost))
				printf("\nCan't transfer minimal cost back to CPU");
			  	d = dtid >> 32;
				count++;
			}
		break;
	}
    
    /*Iterative hill approch */
    // start1 = clock();
	// long dist=best_initial_dst;
	// long dst2=best_initial_dst;
    // float cost=0;
    // float x=0,y=0;
    // register int change=0;
    // count=0;

    // do{
    //     cost=0;
    //     dist=dst2;
    //     for(i=0;i<(cities-1);i++)
    //     {	
    
    //         for(j = i+1; j < cities; j++)
    //         {
    //             cost = dist;			
    //             change = distD(i,j,px,py) 
    //             + distD(i+1,(j+1)%cities,px,py) 
    //             - distD(i,(i+1)%cities,px,py)
    //             - distD(j,(j+1)%cities,px,py);
    //             cost += change;	
    //             if(cost < dst2)
    //             {
    //                 x = i;
    //                 y = j;
    //                 dst2 = cost;
    //             }
    //         }

    //     }
    //     if(dst2<dist)
    //     {
    //         float *tmp_x,*tmp_y;
    //         tmp_x=(float*)malloc(sizeof(float)*(y-x));	
    //         tmp_y=(float*)malloc(sizeof(float)*(y-x));	
    //         for(j=0,i=y;i>x;i--,j++)
    //         {
    //             tmp_x[j]=px[i];
    //             tmp_y[j]=py[i];
    //         }
    //         for(j=0,i=x+1;i<=y;i++,j++)
    //         {
    //             px[i]=tmp_x[j];
    //             py[i]=tmp_y[j];
    //         }
    //         free(tmp_x);
    //         free(tmp_y);
    //     }
    //     count++;
    // }while(dst2<dist);


	printf("\n-------------------------------------------------------------------");
	printf("\nleast initial cost is %d",best_initial_dst);
	printf("\nInitial solution time taken is %f",tm);
	printf("\ninitial start city is %d",best_start_city);
	printf("\nMinimal distance found %ld\n",dst2);
	printf("\nnumber of times hill climbed in minimal distance solution %d\n",count);
	end1 = clock();
	printf("\ntime : %f\n",((double) (end1 - start1)) / CLOCKS_PER_SEC);

	free(posx);
	free(posy);

	free(dst_host);

	hipFree(d_posx);
	hipFree(d_posy);
	hipFree(dst);
	hipFree(visited);
	hipFree(r_device);
	return 0;
}



#include <hip/hip_runtime.h>
#include"stdio.h"
#include <string.h>
#include <stdlib.h>
#include <time.h>
#include"math.h"
#include <ctype.h>
#include <assert.h>

/*this code shows that starting from the least cost initial solution through NN and performing 2 opt only on the best initial solution 
may or may not reach a better solution than IHC.c (varies for instances - gives improvement for 300 and 1000 instances, but not for
instance size 100)*/

/* Euclidean distance calculation */
__device__ long GPU_distD(int i,int j,float *x,float*y)
{
	float dx=x[i]-x[j];
	float dy=y[i]-y[j]; 
	return(sqrtf( (dx*dx) + (dy*dy) ));
}

long distD(int i,int j,float *x,float*y)
{
	float dx=x[i]-x[j];
	float dy=y[i]-y[j]; 
	return(sqrtf( (dx*dx) + (dy*dy) ));
}

// __device__ void routeChecker(long N,int *r)
// {
// 	int *v,i,flag=0;
// 	v=(int*)calloc(N,sizeof(int));	

// 	for(i=0;i<N;i++)
// 		v[r[i]]++;
// 	for(i=0;i<N;i++)
// 	{
// 		if(v[i] != 1 )
// 		{
// 			flag=1;
// 			printf("breaking at %d",i);
// 			break;
// 		}
// 	}
// 	if(flag==1)
// 		printf("\nroute is not valid");
// 	// else
// 	// 	printf("\nroute is valid");
// }

/* Initial solution construction using NN */
__global__ void nn_init(int *route,long cities,float *posx,float*posy,int *visited,long *dst)
{
	int id = threadIdx.x+blockIdx.x*blockDim.x;
	if(id<cities)
	{
		dst[id]=0;
		int start_index=id;
		route[start_index*cities+0]=start_index;
		int k=1,i=start_index,j;
		float min;
		int minj,mini,count=1,flag=0;
		// long dst=0;
		// int *visited=(int*)calloc(cities,sizeof(int));
		visited[start_index*cities+start_index]=1;
		while(count!=cities)
		{
			flag=0;
			for(j=0;j<cities;j++)
			{
				if(i!=j && !visited[start_index*cities+j])
				{
					min=GPU_distD(i,j,posx,posy);
					minj=j;
					break;	
				}
			}

			for(j=minj+1;j<cities;j++)
			{
				
				if( !visited[start_index*cities+j])
				{
					if(min>GPU_distD(i,j,posx,posy))
					{
						min=GPU_distD(i,j,posx,posy);
						mini=j;
						flag=1;				
					}
				}
			}
			if(flag==0)
				i=minj;
			else
				i=mini;
			dst[id]+=min;
			route[start_index*cities+k++]=i;
			visited[start_index*cities+i]=1;
			count++;
		}
		// free(visited);
		dst[id]+=GPU_distD(route[start_index*cities+0],route[start_index*cities+cities-1],posx,posy);
		// routeChecker(cities, route);
	}

	
}


/* Arrange coordinate in initial solution's order*/
void setCoord(int *r,float *posx,float *posy,float *px,float *py,long cities)
{
	int i;
	for(i=0;i<cities;i++)
	{
		px[i]=posx[r[i]];
		py[i]=posy[r[i]];
	}
}

long distH(float *px,float *py,long cit)
{
	float dx,dy;
	long cost=0;
	int i;
	for(i=0;i<(cit-1);i++)
	{
		dx=px[i]-px[i+1];
		dy=py[i]-py[i+1]; 
		cost+=sqrtf( (dx*dx) + (dy*dy) );
	}
	dx=px[i]-px[0];
	dy=py[i]-py[0]; 
	cost+=sqrtf( (dx*dx) + (dy*dy) );
	return cost;

}

int minn(int a,int b)
{
	if(a<b)
	{
		return a;
	}
	return b;
}

int main(int argc, char *argv[])
{
	int ch, cnt, in1;
	float in2, in3;
	FILE *f;
	float *posx, *posy;
	float *px, *py,tm;
	char str[256];  
	int *r;
	long sol,d,cities,no_pairs,tid=0;
	int i,j,intl,count;
	
	clock_t start,end,start1,end1;

	f = fopen(argv[1], "r");
	if (f == NULL) {fprintf(stderr, "could not open file \n");  exit(-1);}

	ch = getc(f);  while ((ch != EOF) && (ch != '\n')) ch = getc(f);
	ch = getc(f);  while ((ch != EOF) && (ch != '\n')) ch = getc(f);
	ch = getc(f);  while ((ch != EOF) && (ch != '\n')) ch = getc(f);

	ch = getc(f);  while ((ch != EOF) && (ch != ':')) ch = getc(f);
	fscanf(f, "%s\n", str);
	cities = atoi(str);
	if (cities <= 2) {fprintf(stderr, "only %ld cities\n", cities);  exit(-1);}

	sol=cities*(cities-1)/2;
	posx = (float *)malloc(sizeof(float) * cities);  if (posx == NULL) {fprintf(stderr, "cannot allocate posx\n");  exit(-1);}
	posy = (float *)malloc(sizeof(float) * cities);  if (posy == NULL) {fprintf(stderr, "cannot allocate posy\n");  exit(-1);}
	px = (float *)malloc(sizeof(float) * cities);  if (px == NULL) {fprintf(stderr, "cannot allocate posx\n");  exit(-1);}
	py = (float *)malloc(sizeof(float) * cities);  if (py == NULL) {fprintf(stderr, "cannot allocate posy\n");  exit(-1);}
	
	r = (int *)malloc(sizeof(int) * cities);
	ch = getc(f);  while ((ch != EOF) && (ch != '\n')) ch = getc(f);
	fscanf(f, "%s\n", str);
	if (strcmp(str, "NODE_COORD_SECTION") != 0) {fprintf(stderr, "wrong file format\n");  exit(-1);}

	cnt = 0;

	while (fscanf(f, "%d %f %f\n", &in1, &in2, &in3)) 
	{
		posx[cnt] = in2;
		posy[cnt] = in3;
		cnt++;
		if (cnt > cities) {fprintf(stderr, "input too long\n");  exit(-1);}
		if (cnt != in1) {fprintf(stderr, "input line mismatch: expected %d instead of %d\n", cnt, in1);  exit(-1);}
	}

	if (cnt != cities) {fprintf(stderr, "read %d instead of %ld cities\n", cnt, cities);  exit(-1);}
	fscanf(f, "%s", str);
	if (strcmp(str, "EOF") != 0) {fprintf(stderr, "didn't see 'EOF' at end of file\n");  exit(-1);}

	// int dst_final=INT_MAX;
	// int count_final;
	// int best_initial_dst=INT_MAX;
	// double best_initial_time;
	// int best_start_city;
    // int*best_initial_route = (int *)malloc(sizeof(int) * cities);

	// if(cities>1000)
	// {
	// 	printf("too many cities, code does not support yet");
	// 	return 0;
	// }

	if(cities<1)
	{
		printf("too less cities");
		return 0;
	}

	long *dst;
	int *visited;
	long *dst_host;
	int *r_device;

	if(hipSuccess!=hipMalloc((void**)&dst,sizeof(long)*cities))
	printf("\nCan't allocate memory for dst in device");

	dst_host=(long*)malloc(sizeof(long)*(cities));	

	if(hipSuccess!=hipMalloc((void**)&visited,sizeof(int)*(cities*cities)))
	printf("\nCan't allocate memory for visited in device");

	if(hipSuccess!=hipMalloc((void**)&r_device,sizeof(int)*(cities*cities)))
	printf("\nCan't allocate memory for r i.e route in device");

	float *d_posx, *d_posy;

	if(hipSuccess!=hipMalloc((void**)&d_posx,sizeof(float)*cities))
	printf("\nCan't allocate memory for coordinate x on GPU");
	if(hipSuccess!=hipMalloc((void**)&d_posy,sizeof(float)*cities))
	printf("\nCan't allocate memory for coordinate y on GPU");

	if(hipSuccess!=hipMemcpy(d_posx,posx,sizeof(float)*cities,hipMemcpyHostToDevice))
	printf("\nCan't transfer px on GPU");
	if(hipSuccess!=hipMemcpy(d_posy,posy,sizeof(float)*cities,hipMemcpyHostToDevice))
	printf("\nCan't transfer py on GPU");

	start = clock();
	
	/*Calling NN algo for initial solution creation*/
	nn_init<<<(cities-1/1024)+1,minn(cities,1024)>>>(r_device,cities,d_posx,d_posy,visited,dst);

	end = clock();

	if(hipSuccess!=hipMemcpy(dst_host,dst,sizeof(long)*cities,hipMemcpyDeviceToHost))
	printf("\nCan't transfer dst values back to CPU");

	tm = ((double) (end - start)) / CLOCKS_PER_SEC;

	long best_initial_dst=INT_MAX;
	int best_start_city;

	for(int i=0;i<cities;i++)
	{
		// printf("\nindex : %d , value at index : %ld",i,dst_host[i]);
		if(dst_host[i]<best_initial_dst)
		{
			best_initial_dst=dst_host[i];
			best_start_city=i;
		}
	}

	int *req_r=r_device+best_start_city*cities; //move only the route which corresponds to minimum initial dst

	if(hipSuccess!=hipMemcpy(r,req_r,sizeof(int)*cities,hipMemcpyDeviceToHost))
	printf("\nCan't transfer best route values back to CPU");

	// for(int i=0;i<cities;i++)
	// {
	// 	printf("\n%d",r[i]);
	// }

    setCoord(r,posx,posy,px,py,cities);
    
    /*Iterative hill approch */
    start1 = clock();
	long dist=best_initial_dst;
	long dst2=best_initial_dst;
    float cost=0;
    float x=0,y=0;
    register int change=0;
    count=0;

    do{
        cost=0;
        dist=dst2;
        for(i=0;i<(cities-1);i++)
        {	
    
            for(j = i+1; j < cities; j++)
            {
                cost = dist;			
                change = distD(i,j,px,py) 
                + distD(i+1,(j+1)%cities,px,py) 
                - distD(i,(i+1)%cities,px,py)
                - distD(j,(j+1)%cities,px,py);
                cost += change;	
                if(cost < dst2)
                {
                    x = i;
                    y = j;
                    dst2 = cost;
                }
            }

        }
        if(dst2<dist)
        {
            float *tmp_x,*tmp_y;
            tmp_x=(float*)malloc(sizeof(float)*(y-x));	
            tmp_y=(float*)malloc(sizeof(float)*(y-x));	
            for(j=0,i=y;i>x;i--,j++)
            {
                tmp_x[j]=px[i];
                tmp_y[j]=py[i];
            }
            for(j=0,i=x+1;i<=y;i++,j++)
            {
                px[i]=tmp_x[j];
                py[i]=tmp_y[j];
            }
            free(tmp_x);
            free(tmp_y);
        }
        count++;
    }while(dst2<dist);


	printf("\n-------------------------------------------------------------------");
	printf("\nleast initial cost is %d",best_initial_dst);
	printf("\ntime taken is %f",tm);
	printf("\ninitial start city is %d",best_start_city);
	printf("\nMinimal distance found %ld\n",dst2);
	printf("\nnumber of times hill climbed in minimal distance solution %d\n",count);
	end1 = clock();
	printf("\ntime : %f\n",((double) (end1 - start1)) / CLOCKS_PER_SEC);

	free(posx);
	free(posy);

	free(dst_host);

	hipFree(d_posx);
	hipFree(d_posy);
	hipFree(dst);
	hipFree(visited);
	hipFree(r_device);
	return 0;
}


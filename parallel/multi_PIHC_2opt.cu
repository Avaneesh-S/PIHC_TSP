
#include <hip/hip_runtime.h>
#include"stdio.h"
#include <string.h>
#include <stdlib.h>
#include <time.h>
#include"math.h"
#include <ctype.h>
#include <assert.h>

/*code to perform 2opt on every initial solution, that is construct initial solution with every city as start city and run 2 opt on 
all in parallel*/

/* Euclidean distance calculation */
__host__ __device__ long distD(int i,int j,float *x,float*y)
{
	float dx=x[i]-x[j];
	float dy=y[i]-y[j]; 
	return(sqrtf( (dx*dx) + (dy*dy) ));
}

// long distD(int i,int j,float *x,float*y)
// {
// 	float dx=x[i]-x[j];
// 	float dy=y[i]-y[j]; 
// 	return(sqrtf( (dx*dx) + (dy*dy) ));
// }

// __device__ void routeChecker(long N,int *r)
// {
// 	int *v,i,flag=0;
// 	v=(int*)calloc(N,sizeof(int));	

// 	for(i=0;i<N;i++)
// 		v[r[i]]++;
// 	for(i=0;i<N;i++)
// 	{
// 		if(v[i] != 1 )
// 		{
// 			flag=1;
// 			printf("breaking at %d",i);
// 			break;
// 		}
// 	}
// 	if(flag==1)
// 		printf("\nroute is not valid");
// 	// else
// 	// 	printf("\nroute is valid");
// }

/* Initial solution construction using NN */
__global__ void nn_init(int *route,long cities,float *posx,float*posy,int *visited,long *dst,unsigned long long *dst_tid)
{
	int id = threadIdx.x+blockIdx.x*blockDim.x;
	if(id<cities)
	{
		dst[id]=0;
		int start_index=id;
		route[start_index*cities+0]=start_index;
		int k=1,i=start_index,j;
		float min;
		int minj,mini,count=1,flag=0;
		// long dst=0;
		// int *visited=(int*)calloc(cities,sizeof(int));
		visited[start_index*cities+start_index]=1;
		while(count!=cities)
		{
			flag=0;
			for(j=0;j<cities;j++)
			{
				if(i!=j && !visited[start_index*cities+j])
				{
					min=distD(i,j,posx,posy);
					minj=j;
					break;	
				}
			}

			for(j=minj+1;j<cities;j++)
			{
				
				if( !visited[start_index*cities+j])
				{
					if(min>distD(i,j,posx,posy))
					{
						min=distD(i,j,posx,posy);
						mini=j;
						flag=1;				
					}
				}
			}
			if(flag==0)
				i=minj;
			else
				i=mini;
			dst[id]+=min;
			route[start_index*cities+k++]=i;
			visited[start_index*cities+i]=1;
			count++;
		}
		// free(visited);
		dst[id]+=distD(route[start_index*cities+0],route[start_index*cities+cities-1],posx,posy);
		dst_tid[id]=(((long)dst[id]+1) << 32) -1;
		// routeChecker(cities, route);
	}

	
}


/* Arrange coordinate in initial solution's order*/
__global__ void setCoord(int *r,float *posx,float *posy,float *px,float *py,long cities)
{
	int id= threadIdx.x+blockIdx.x*blockDim.x;
	if(id<cities)
	{
		int i;
		for(i=id*cities;i<id*cities+cities;i++)
		{
			px[i]=posx[r[i]];
			py[i]=posy[r[i]];
		}
	}
}

long distH(float *px,float *py,long cit)
{
	float dx,dy;
	long cost=0;
	int i;
	for(i=0;i<(cit-1);i++)
	{
		dx=px[i]-px[i+1];
		dy=py[i]-py[i+1]; 
		cost+=sqrtf( (dx*dx) + (dy*dy) );
	}
	dx=px[i]-px[0];
	dy=py[i]-py[0]; 
	cost+=sqrtf( (dx*dx) + (dy*dy) );
	return cost;

}

int minn(int a,int b)
{
	if(a<b)
	{
		return a;
	}
	return b;
}

/*A kenel function that finds a minimal weighted neighbor using TPR mapping strategy*/
__global__ void tsp_tpr(float *pox,float *poy,long *initcost,unsigned long long *dst_tid,long cit)
{

	long id,j;
	long i=threadIdx.x+blockIdx.x*blockDim.x;
	register long change,mincost=initcost[i%cit],cost;
	if(i < cit*(cit-1))
	{
		long limit = ((long)(i/cit)*cit)+cit;
		for(j=i+1;j<limit;j++)
		{
			change = 0; cost=initcost[i%cit];
			change=distD(i,j,pox,poy)+distD((i+1)%(cit*cit),(j+1)%(cit*cit),pox,poy)-distD(i,(i+1)%(cit*cit),pox,poy)-distD(j,(j+1)%(cit*cit),pox,poy);
			cost+=change;	
			if(cost < mincost)
			{
				mincost = cost;
				id = i%cit * (cit-1)+(j%cit-1)-i%cit*(i%cit+1)/2;	
			}	 

		}
		if(mincost < initcost[i%cit])
			 atomicMin(dst_tid+(i%cit), ((unsigned long long)mincost << 32) | id);

	}
	
}



/* At each IHC steps, XY coordinates are arranged using next initial solution's order*/
void twoOpt(long x,long y,float *pox,float *poy)
{
	float *tmp_x,*tmp_y;
	int i,j;
	tmp_x=(float*)malloc(sizeof(float)*(y-x));	
	tmp_y=(float*)malloc(sizeof(float)*(y-x));
	for(j=0,i=y;i>x;i--,j++)
	{
		tmp_x[j]=pox[i];
		tmp_y[j]=poy[i];
	}
	for(j=0,i=x+1;i<=y;i++,j++)
	{
		pox[i]=tmp_x[j];
		poy[i]=tmp_y[j];
	}
	free(tmp_x);
	free(tmp_y);

}

int main(int argc, char *argv[])
{
	int ch, cnt, in1;
	float in2, in3;
	FILE *f;
	float *posx, *posy;
	float *px, *py,tm;
	char str[256];  
	int *r;
	long sol,cities,no_pairs;
	int i,j,intl,count;
	
	clock_t start,end,start1,end1;

	f = fopen(argv[1], "r");
	if (f == NULL) {fprintf(stderr, "could not open file \n");  exit(-1);}

	ch = getc(f);  while ((ch != EOF) && (ch != '\n')) ch = getc(f);
	ch = getc(f);  while ((ch != EOF) && (ch != '\n')) ch = getc(f);
	ch = getc(f);  while ((ch != EOF) && (ch != '\n')) ch = getc(f);

	ch = getc(f);  while ((ch != EOF) && (ch != ':')) ch = getc(f);
	fscanf(f, "%s\n", str);
	cities = atoi(str);
	if (cities <= 2) {fprintf(stderr, "only %ld cities\n", cities);  exit(-1);}

	sol=cities*(cities-1)/2;
	posx = (float *)malloc(sizeof(float) * cities);  if (posx == NULL) {fprintf(stderr, "cannot allocate posx\n");  exit(-1);}
	posy = (float *)malloc(sizeof(float) * cities);  if (posy == NULL) {fprintf(stderr, "cannot allocate posy\n");  exit(-1);}
	px = (float *)malloc(sizeof(float) * (cities*cities));  if (px == NULL) {fprintf(stderr, "cannot allocate posx\n");  exit(-1);}
	py = (float *)malloc(sizeof(float) * (cities*cities));  if (py == NULL) {fprintf(stderr, "cannot allocate posy\n");  exit(-1);}
	
	r = (int *)malloc(sizeof(int) * cities);
	ch = getc(f);  while ((ch != EOF) && (ch != '\n')) ch = getc(f);
	fscanf(f, "%s\n", str);
	if (strcmp(str, "NODE_COORD_SECTION") != 0) {fprintf(stderr, "wrong file format\n");  exit(-1);}

	cnt = 0;

	while (fscanf(f, "%d %f %f\n", &in1, &in2, &in3)) 
	{
		posx[cnt] = in2;
		posy[cnt] = in3;
		cnt++;
		if (cnt > cities) {fprintf(stderr, "input too long\n");  exit(-1);}
		if (cnt != in1) {fprintf(stderr, "input line mismatch: expected %d instead of %d\n", cnt, in1);  exit(-1);}
	}

	if (cnt != cities) {fprintf(stderr, "read %d instead of %ld cities\n", cnt, cities);  exit(-1);}
	fscanf(f, "%s", str);
	if (strcmp(str, "EOF") != 0) {fprintf(stderr, "didn't see 'EOF' at end of file\n");  exit(-1);}


	if(cities<1)
	{
		printf("too less cities");
		return 0;
	}

	long *dst;
	unsigned long long *d_dst_tid;
	int *visited;
	long *dst_host;
	int *r_device;

	if(hipSuccess!=hipMalloc((void**)&dst,sizeof(long)*cities))
	printf("\nCan't allocate memory for dst in device");

	if(hipSuccess!=hipMalloc((void**)&d_dst_tid,sizeof(unsigned long long)*cities))
	printf("\nCan't allocate memory for dst_id in device");

	dst_host=(long*)malloc(sizeof(long)*(cities));	

	if(hipSuccess!=hipMalloc((void**)&visited,sizeof(int)*(cities*cities)))
	printf("\nCan't allocate memory for visited in device");

	if(hipSuccess!=hipMalloc((void**)&r_device,sizeof(int)*(cities*cities)))
	printf("\nCan't allocate memory for r i.e route in device");

	float *d_posx, *d_posy;

	if(hipSuccess!=hipMalloc((void**)&d_posx,sizeof(float)*cities))
	printf("\nCan't allocate memory for coordinate x on GPU");
	if(hipSuccess!=hipMalloc((void**)&d_posy,sizeof(float)*cities))
	printf("\nCan't allocate memory for coordinate y on GPU");

	if(hipSuccess!=hipMemcpy(d_posx,posx,sizeof(float)*cities,hipMemcpyHostToDevice))
	printf("\nCan't transfer px on GPU");
	if(hipSuccess!=hipMemcpy(d_posy,posy,sizeof(float)*cities,hipMemcpyHostToDevice))
	printf("\nCan't transfer py on GPU");

	float *d_px, *d_py;
	if(hipSuccess!=hipMalloc((void**)&d_px,sizeof(float)*(cities*cities)))
	printf("\nCan't allocate memory for coordinate x on GPU");
	if(hipSuccess!=hipMalloc((void**)&d_py,sizeof(float)*(cities*cities)))
	printf("\nCan't allocate memory for coordinate y on GPU");


	start = clock();
	
	/*Calling NN algo for initial solution creation*/
	nn_init<<<(cities-1/1024)+1,minn(cities,1024)>>>(r_device,cities,d_posx,d_posy,visited,dst,d_dst_tid);
	// cudaDeviceSynchronize();

	if(hipSuccess!=hipMemcpy(dst_host,dst,sizeof(long)*cities,hipMemcpyDeviceToHost))
	printf("\nCan't transfer dst values back to CPU");

	end = clock();

	tm = ((double) (end - start)) / CLOCKS_PER_SEC;

	long least_dst=LONG_MAX;
	// int best_start_city;

	for(int itr=0;itr<cities;itr++)
	{
		// printf("\nindex : %d , value at index : %ld",i,dst_host[i]);
		if(dst_host[itr]<least_dst)
		{
			least_dst=dst_host[itr];
			// best_start_city=i;
		}
	}

	printf("\nNN running complete");
	free(posx);
	free(posy);

	free(dst_host);

	// int *req_r=r_device+best_start_city*cities; //move only the route which corresponds to minimum initial dst

	// if(cudaSuccess!=cudaMemcpy(r,req_r,sizeof(int)*cities,cudaMemcpyDeviceToHost))
	// printf("\nCan't transfer best route values back to CPU");

    setCoord<<<(cities-1/1024)+1,minn(cities,1024)>>>(r_device,d_posx,d_posy,d_px,d_py,cities);

	if(hipSuccess!=hipMemcpy(px,d_px,sizeof(float)*(cities*cities),hipMemcpyDeviceToHost))
	printf("\nCan't transfer px values back to CPU");

	if(hipSuccess!=hipMemcpy(py,d_py,sizeof(float)*(cities*cities),hipMemcpyDeviceToHost))
	printf("\nCan't transfer py values back to CPU");

	printf("\ninitial solution part done");

	int blk,thrd;
	// unsigned long long *d_dst_tid;
	// long dst2=best_initial_dst;

	start1 = clock();
	count = 1;
	// unsigned long long dst_tid = (((long)dst2+1) << 32) -1;
	long itr=floor(cities/2);
	int nx, ny;
	if(cities <= 32)
	{
		blk = 1 ;
		nx = cities;
		ny = cities;
	}
	else
	{
		blk = (cities - 1) / 32 + 1;
		nx = 32;
		ny = 32;
	}
	dim3 thrds (nx,ny);
	dim3 blks (blk,blk);

	unsigned long long *dtid=(unsigned long long*)malloc(sizeof(unsigned long long)*(cities));
	long *tid=(long*)malloc(sizeof(long)*(cities));	
	long *d=(long*)malloc(sizeof(long)*(cities));
	long min_d=LONG_MAX;


	blk=((cities*(cities-1)-1)/1024+1);
	thrd=1024;
	
	
	
	
	tsp_tpr<<<blk,thrd>>>(d_px,d_py,dst,d_dst_tid,cities);
	
	if(hipSuccess!=hipMemcpy(dtid,d_dst_tid,sizeof(unsigned long long)*cities,hipMemcpyDeviceToHost))
	printf("\nCan't transfer minimal dtid to CPU");

	printf("\ntpr finished running");

	for(int itr=0;itr<cities;itr++)
	{
		d[itr] = dtid[itr] >> 32;
		if(d[itr]<min_d)
		{
			min_d=d[itr];
		}
	}

	printf("\n first tpr call complete moved min d");
	long *x=(long*)malloc(sizeof(long)*(cities));
	long *y=(long*)malloc(sizeof(long)*(cities));
	
	
	while( min_d < least_dst )
	{
		least_dst=min_d;
		for(int itr=0;itr<cities;itr++)
		{
			tid[itr] = dtid[itr] & ((1ull<<32)-1);
			x[itr]=cities-2-floor((sqrt(8*(sol-tid[itr]-1)+1)-1)/2);
			y[itr]=tid[itr]-x[itr]*(cities-1)+(x[itr]*(x[itr]+1)/2)+1;
			twoOpt(x[itr],y[itr],px+(cities*itr),py+(cities*itr));
			if(hipSuccess!=hipMemcpy(d_px+(itr*cities),px+(cities*itr),sizeof(float)*cities,hipMemcpyHostToDevice))
			printf("\nCan't transfer px on GPU");
			if(hipSuccess!=hipMemcpy(d_py+(itr*cities),py+(cities*itr),sizeof(float)*cities,hipMemcpyHostToDevice))
			printf("\nCan't transfer py on GPU");
			// unsigned long long dst_tid = (((long)least_dst+1) << 32) -1;
			// if(cudaSuccess!=cudaMemcpy(d_dst_tid[itr],&dst_tid,sizeof(unsigned long long),cudaMemcpyHostToDevice))
			// printf("\nCan't transfer dst_tid on GPU");
		} 

		tsp_tpr<<<blk,thrd>>>(d_px,d_py,dst,d_dst_tid,cities);

		if(hipSuccess!=hipMemcpy(dtid,d_dst_tid,sizeof(unsigned long long)*cities,hipMemcpyDeviceToHost))
		printf("\nCan't transfer minimal dtid to CPU inside loop");


		for(int itr=0;itr<cities;itr++)
		{
			d[itr] = dtid[itr] >> 32;
			if(d[itr]<min_d)
			{
				min_d=d[itr];
			}
		}
		count++;
	}


	printf("\n-------------------------------------------------------------------");
	// printf("\nleast initial cost is %d",best_initial_dst);
	printf("\nInitial solution time taken is %f",tm);
	// printf("\ninitial start city is %d",best_start_city);
	printf("\nMinimal distance found %ld\n",min_d);
	printf("\nnumber of times hill climbed in minimal distance solution %d\n",count);
	end1 = clock();
	printf("\ntime : %f\n",((double) (end1 - start1)) / CLOCKS_PER_SEC);


	free(x);
	free(y);
	free(tid);
	free(d);

	hipFree(d_posx);
	hipFree(d_posy);
	hipFree(dst);
	hipFree(visited);
	hipFree(r_device);
	hipFree(d_dst_tid);
	
	return 0;
}

